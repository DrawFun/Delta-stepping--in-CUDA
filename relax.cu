#include "hip/hip_runtime.h"
#ifndef _RELAX_H_
#define _RELAX_H_
#include "vertex.h"

__global__ void
get_result(cpu::vertex* gpu_global_vertex,int i){
	printf("result: %d\n",gpu_global_vertex[i].dist);	
}

__global__ void
relax_all(int* gpu_vertex_buf, cpu::gpuResult* gpu_used_result_buf,
	       cpu::vertex* gpu_global_vertex, cpu::edge* gpu_global_edge){

    const unsigned int bid = blockIdx.x; 
    const unsigned int num_block = gridDim.x; 
    const unsigned int tid_in_block = threadIdx.x;
    const unsigned int num_thread = blockDim.x;
    const unsigned int tid_in_grid = blockDim.x * blockIdx.x +threadIdx.x;

    int i=0,j=0;
    int dist_current,dest,tent_dest;
    __shared__ int result_count,lock;
    if(tid_in_block==0){
	result_count=0;
	lock=0;
    }

    //one vertex per block
    for (i=bid;i<V_BUF_SIZE;i+=num_block){

        if(gpu_vertex_buf[i] == 0)
            return;

	//get current vertex's info
        //cpu::vertex *temp_v = &gpu_global_vertex[gpu_vertex_buf[i]];
	int edge_index = gpu_global_vertex[gpu_vertex_buf[i]].edge_index;
	cpu::gpuResult *current_result_buf = &gpu_used_result_buf[bid*MAX_RESULT_SIZE]; //the buffer now used
        int num_edges = gpu_global_vertex[gpu_vertex_buf[i]+1].edge_index - edge_index;
        int tent_current = gpu_global_vertex[gpu_vertex_buf[i]].dist;

	//one edge per thread
        for(j=tid_in_block;j<num_edges;j+=num_thread){
		//get edge's info
                dist_current = gpu_global_edge[edge_index+j].distance;
                dest = gpu_global_edge[edge_index+j].des_v;
                tent_dest = gpu_global_vertex[dest].dist;
            if(tent_current + dist_current > MAX_DISTANCE)
                printf("!!!!!!\n");
            if(tent_current + dist_current < tent_dest){
                gpu_global_vertex[dest].dist = tent_current + dist_current;
                  
	    //FIXME: bad critical section
	    int now,loop=0;
while(loop==0){
if(atomicExch(&lock,1)==0){
	    now = result_count;
	    atomicAdd(&result_count,1);
	    loop=1;
	    atomicExch(&lock,0);
	    }
}
		current_result_buf[now].index = dest;
            	current_result_buf[now].old_distance = tent_dest;
            	current_result_buf[now].new_distance = (tent_current+dist_current);
//printf("%d %d %d\n",dest,tent_dest,tent_current+dist_current);
//printf("GPU:%d->%d old:%d new:%d %d %d\n",gpu_vertex_buf[i],current_result_buf[now].index,current_result_buf[now].old_distance,current_result_buf[now].new_distance,now,result_count);
        }
	}
    }
 }

#endif
