#include "hip/hip_runtime.h"
#ifndef _RELAX_H_
#define _RELAX_H_
#include "vertex.h"

__global__ void
get_result(cpu::vertex* gpu_global_vertex,int des,int src){
	printf("result: %d\n",gpu_global_vertex[des].dist);	
	int pre = gpu_global_vertex[des].pre_vertex;
	printf("(%d,%d)",des,gpu_global_vertex[des].dist);
	while(pre != src){
	   printf(" <- (%d,%d)",pre,gpu_global_vertex[pre].dist);
	   pre = gpu_global_vertex[pre].pre_vertex;
	}
	printf(" <- (%d,%d)\n",src,gpu_global_vertex[src].dist);
}

__global__ void
verify_result(cpu::vertex* gpu_global_vertex,cpu::gpuResult *gpu_result){

    const unsigned int tid = threadIdx.x;

//    for(int i=0;i<NUM_BLOCK;i++){
       int count=0;
       cpu::gpuResult* current = &gpu_result[tid*MAX_RESULT_SIZE];
       while(1){
       if(current[count].index==0)
          break;
       	  if(current[count].new_distance<gpu_global_vertex[current[count].index].dist){
	  	    //printf("VERIFY!!!!\n");
		    gpu_global_vertex[current[count].index].dist = current[count].new_distance;
		    gpu_global_vertex[current[count].index].pre_vertex = current[count].pre;
       	  }
       count++;
       }
//    }
}

__global__ void
relax_all(int* gpu_vertex_buf, cpu::gpuResult* gpu_used_result_buf,
	       cpu::vertex* gpu_global_vertex, cpu::edge* gpu_global_edge){

    const unsigned int bid = blockIdx.x; 
    const unsigned int num_block = gridDim.x; 
    const unsigned int tid_in_block = threadIdx.x;
    const unsigned int num_thread = blockDim.x;
    const unsigned int tid_in_grid = blockDim.x * blockIdx.x +threadIdx.x;

    int i=0,j=0;
    int dist_current,dest,tent_dest;
    __shared__ int result_count,lock;
    if(tid_in_block==0){
	result_count=0;
	lock=0;
    }

    //one vertex per block
    for (i=bid;i<V_BUF_SIZE;i+=num_block){

        if(gpu_vertex_buf[i] == 0)
            return;

	//get current vertex's info
        //cpu::vertex *temp_v = &gpu_global_vertex[gpu_vertex_buf[i]];
	int edge_index = gpu_global_vertex[gpu_vertex_buf[i]].edge_index;
	cpu::gpuResult *current_result_buf = &gpu_used_result_buf[bid*MAX_RESULT_SIZE]; //the buffer now used
        int num_edges = gpu_global_vertex[gpu_vertex_buf[i]+1].edge_index - edge_index;
        int tent_current = gpu_global_vertex[gpu_vertex_buf[i]].dist;

	//one edge per thread
        for(j=tid_in_block;j<num_edges;j+=num_thread){
		//get edge's info
            dist_current = gpu_global_edge[edge_index+j].distance;
            dest = gpu_global_edge[edge_index+j].des_v;
            tent_dest = gpu_global_vertex[dest].dist;

            //if(tent_current + dist_current > MAX_DISTANCE)
                //printf("DISTANCE BOOM\n");

            if(tent_current + dist_current < gpu_global_vertex[dest].dist){
                gpu_global_vertex[dest].dist = tent_current + dist_current;
		        gpu_global_vertex[dest].pre_vertex = gpu_vertex_buf[i];
                  
	            //FIXME: bad critical section
        	    int now,loop=0;

                while(loop==0){
                    if(atomicExch(&lock,1)==0){
            	        now = result_count;
            	        atomicAdd(&result_count,1);
                	    loop=1;
                	    atomicExch(&lock,0);
        	        }
                }
        		current_result_buf[now].index = dest;
            	current_result_buf[now].old_distance = tent_dest;
            	current_result_buf[now].new_distance = (tent_current+dist_current);
		        current_result_buf[now].pre = gpu_vertex_buf[i];
                //printf("%d %d %d\n",dest,tent_dest,tent_current+dist_current);
                //printf("GPU:%d->%d old:%d new:%d %d %d\n",gpu_vertex_buf[i],current_result_buf[now].index,current_result_buf[now].old_distance,current_result_buf[now].new_distance,now,result_count);
        }
	}
    }
 }



#endif
