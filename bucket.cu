#include "hip/hip_runtime.h"
//#include "cutil.h"
#include "cutil_inline.h"
//#include "vertex.h"
#include "relax.cu"
#define DEBUG
#ifdef DEBUG
#define dprintf(x...) printf(x)
#else 
#define dprintf(x...)
#endif

int main(void){
    int num_block = 8;
    int num_threads = 32;
    dim3 dg(num_block, 1, 1);
    dim3 db(num_threads, 1, 1);
    cpu cpu_instance("hi.gr");
    cpu::gpuResult *gpu_used_result_buf;

    hipSetDevice(cutGetMaxGflopsDeviceId());

     //copy to GPU
    CUDA_SAFE_CALL(hipMalloc((void **)&cpu_instance.gpu_vertex,(cpu_instance.vertex_size+2)*sizeof(cpu::vertex)));
    CUDA_SAFE_CALL(hipMalloc((void**)&cpu_instance.gpu_edge,cpu_instance.edges_size*sizeof(cpu::edge)));
    //malloc danteng!!
    CUDA_SAFE_CALL(hipMalloc((void **)&gpu_used_result_buf,sizeof(cpu::gpuResult)*num_block*num_threads));  
    //copy  
    CUDA_SAFE_CALL(hipMemcpy(cpu_instance.gpu_vertex,cpu_instance.global_vertex,(cpu_instance.vertex_size+2)*sizeof(cpu::vertex),hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(cpu_instance.gpu_edge,cpu_instance.global_edge,cpu_instance.edges_size*sizeof(cpu::edge), hipMemcpyHostToDevice));

    //malloc vertex buffer
    CUDA_SAFE_CALL(hipMalloc((void**)&cpu_instance.vertex_buf_ptr, MAX_BUCKET_SIZE));
    //malloc result buffer

    int min;
    int temp_vertex_array[MAX_BUCKET_SIZE];
    
    while(!cpu_instance.is_all_bucket_empty()){
        min = cpu_instance.min_no_empty_bucket(); 
        int count = cpu_instance.bucket_set_to_array(min, temp_vertex_array);
        //FIXME, send count may have better performance
        for(int i=count; i< MAX_BUCKET_SIZE; i++)
                temp_vertex_array[i] = -1;
        //Copy to CPU-GPU buffer
        CUDA_SAFE_CALL(hipMemcpy(cpu_instance.vertex_buf_ptr,temp_vertex_array, MAX_BUCKET_SIZE,hipMemcpyHostToDevice));
       //call cuda function
       relax_all<<<dg,db>>>(cpu_instance.vertex_buf_ptr, cpu_instance.gpu_vertex,cpu_instance.gpu_edge,cpu_instance.gpu_result_buf,gpu_used_result_buf);
       //relax_all<<<dg,db>>>(cpu_instance.vertex_buf_ptr, cpu_instance.gpu_vertex,cpu_instance.gpu_edge,cpu_instance.gpu_result_buf);
       CUDA_SAFE_CALL(hipMemcpy(cpu_instance.gpu_result_buf,gpu_used_result_buf, sizeof(cpu::gpuResult)*num_threads*num_block, hipMemcpyDeviceToHost)); 
       //get the result from gpu
       int result_count = 0;
       while(1){
            if(result_count >= MAX_BUCKET_SIZE){
                break;
            }
            if(cpu_instance.gpu_result_buf[result_count].index == 0){
                result_count++;
                continue;
             }
            
            int old_index = cpu_instance.gpu_result_buf[result_count].old_distance / cpu_instance.delta;
            int new_index = cpu_instance.gpu_result_buf[result_count].new_distance / cpu_instance.delta;
            cpu_instance.bucket_array[old_index].erase(cpu_instance.gpu_result_buf[result_count].index);
            cpu_instance.bucket_array[new_index].insert(cpu_instance.gpu_result_buf[result_count].index);
            result_count++;
        }
       
    }


    CUDA_SAFE_CALL(hipFree(cpu_instance.gpu_vertex));
    CUDA_SAFE_CALL(hipFree(cpu_instance.gpu_edge));
    cutilSafeCall(hipFree(gpu_used_result_buf));  
    free(cpu_instance.global_vertex);
    free(cpu_instance.global_edge);
    
}

cpu::cpu(char* filepath){
    init_memory(filepath);
    init_all_bucket();
}

cpu::~cpu(){
    free(global_vertex);
    free(global_edge);
}

int cpu::init_graph(){
  int i;
 //vertex size and edge size
  global_vertex =(struct vertex*) malloc((vertex_size+2)*sizeof(struct vertex));
  global_edge = (struct edge*)malloc(edges_size*sizeof(struct edge));

  //init vertex
  for(i=0;i<vertex_size+2;i++){
    global_vertex[i].edges =0;
    global_vertex[i].dist = MAX_DISTANCE;
    global_vertex[i].pre_vertex = -1;
  }

  graph_init=1;
  return 0;
}

int cpu::init_memory(char* filepath){
  char string[256];

  FILE* fp = fopen(filepath,"r");
  if(fp==NULL)
    return -1;

  //init edge, file staff
  while(fgets(string,256,fp)!=NULL){
    static char sign;
    //assumption1: the node start at index 1
    static int src,dest,dist,cur_v=0,cur_edge=0;
    //get the sign of the file line
    sscanf(string,"%c",&sign);

    //the line describe the edge
    if(sign=='a'){
      if(!graph_init)
	return -2;
      if(cur_edge>edges_size)
	return -4;
      sscanf(string,"%c\t%d\t%d\t%d",&sign,&src,&dest,&dist);
      dprintf("edge from:%d to:%d dist:%d\n",src,dest,dist);
      //add the edge to the edge list
      global_edge[cur_edge].des_v=dest;
      global_edge[cur_edge].distance=dist;
      cur_edge++;


      if(cur_v!=src){
	//assumption2: sorted vertex and there is no isolated vertex
	if(cur_v==src-1){
	  global_vertex[src].edges=&global_edge[cur_edge-1];
	  cur_v=src;
	}
	else
	  return -3;
      }
    }
    //the line describe the size of graph
    else if(sign=='p'){ 
      sscanf(string,"%c\tsp\t%d\t%d",&sign,&src,&dest);
      vertex_size = src;
      edges_size = dest;
      #ifdef DEBUG
      dprintf("GOT the size of graph, vertex:%d edge:%d\n",vertex_size,edges_size);
      #endif
      init_graph();
    }
  }

  fclose(fp);
  //copy to GPU
 /* CUDA_SAFE_CALL(hipMalloc((void **)&gpu_vertex,(vertex_size+2)*sizeof(struct vertex)));
  CUDA_SAFE_CALL(hipMalloc((void**)&gpu_edge,edges_size*sizeof(struct edge)));
  CUDA_SAFE_CALL(hipMemcpy(gpu_vertex,global_vertex,(vertex_size+2)*sizeof(struct vertex),hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(gpu_edge,global_edge,edges_size*sizeof(struct edge)));
  */
  return 0;
}

int cpu::init_all_bucket(){
    bucket_array[0].insert(src);
}

int cpu::is_all_bucket_empty(){
    return min_no_empty_bucket()==-1;
}

int cpu::min_no_empty_bucket(){
    for(int i=0;i<MAX_BUKET_NUM;i++){
        if(!bucket_array[i].empty()){
              return i;
        }
    }
    return -1;
}

int cpu::bucket_set_to_array(int index, int* array){
    int count = 0;
    std::set<int>::iterator it = bucket_array[index].begin();
    for(;it!=bucket_array[index].end();it++){
            array[count]=*it;
            count++;
        }
    return count;
}

