#include "hip/hip_runtime.h"
#include "cutil_inline.h"
#include <map>

//#define DEBUG
#include "relax.cu"
#ifdef DEBUG
#define dprintf(x...) printf(x)
#else 
#define dprintf(x...)
#endif


void gpu_memory_prep(cpu &cpu_instance){
     //malloc in GPU
    CUDA_SAFE_CALL(hipMalloc((void **)&cpu_instance.gpu_vertex,(cpu_instance.vertex_size+2)*sizeof(cpu::vertex)));
    CUDA_SAFE_CALL(hipMalloc((void**)&cpu_instance.gpu_edge,cpu_instance.edges_size*sizeof(cpu::edge)));

    //copy to GPU
    CUDA_SAFE_CALL(hipMemcpy(cpu_instance.gpu_vertex,cpu_instance.global_vertex,
			(cpu_instance.vertex_size+2)*sizeof(cpu::vertex),hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(cpu_instance.gpu_edge,cpu_instance.global_edge,
			cpu_instance.edges_size*sizeof(cpu::edge), hipMemcpyHostToDevice));

    //alloc 2 buffer in GPU
    int vertex_buf_size = V_BUF_SIZE * sizeof(int);
    CUDA_SAFE_CALL(hipMalloc((void **)&cpu_instance.gpu_vertex_buf,vertex_buf_size));
    int result_size = MAX_RESULT_SIZE * sizeof(cpu::gpuResult)*NUM_BLOCK;
    CUDA_SAFE_CALL(hipMalloc((void **)&cpu_instance.gpu_used_result_buf,result_size));

    //alloc cpu memory
    cpu_instance.vertex_buf_ptr =(int *) malloc(vertex_buf_size);
    cpu_instance.gpu_result_buf =(cpu::gpuResult *) malloc(result_size);

    //initial gpu_used result buffer
    CUDA_SAFE_CALL(hipMemset(cpu_instance.gpu_used_result_buf,0,result_size));
}

void profile_result(cpu::gpuResult *ptr){
     for(int j=0;j<MAX_RESULT_SIZE;j++)
        printf(" %d ",ptr[j].index);
}

void parse_result(cpu &cpu_instance){
     int result_count = 0;
     //std::map<int,int> leak;

     for(int i=0;i<NUM_BLOCK;i++){
       cpu::gpuResult *current_result = &cpu_instance.gpu_result_buf[i*MAX_RESULT_SIZE];
       result_count=0;
       while(1){
            if(result_count >= MAX_RESULT_SIZE){
                break;
            }
            if(current_result[result_count].index == 0){
                result_count++;
                break;//continue;
            }
            int old_index = current_result[result_count].old_distance / cpu_instance.delta;
            int new_index = current_result[result_count].new_distance / cpu_instance.delta;
            if(current_result[result_count].old_distance != MAX_DISTANCE){
                cpu_instance.bucket_array[old_index].erase(current_result[result_count].index);
	    }
            cpu_instance.bucket_array[new_index].insert(current_result[result_count].index);
            result_count++;
        }
     }
}
    int relax_time =0;
void cal_shortest_path(cpu &cpu_instance){

    int num_block = NUM_BLOCK;
    int num_threads = 32;
    dim3 dg(num_block, 1, 1);
    dim3 db(num_threads, 1, 1);

    int min;
    int vertex_buf_size = V_BUF_SIZE * sizeof(int);
    int result_size = MAX_RESULT_SIZE*sizeof(cpu::gpuResult)*NUM_BLOCK;


   while(!cpu_instance.is_all_bucket_empty()){

        min = cpu_instance.min_no_empty_bucket();



	    memset(cpu_instance.vertex_buf_ptr,0,vertex_buf_size);
	    CUDA_SAFE_CALL(hipMemset(cpu_instance.gpu_used_result_buf,0,result_size));

	    //copy&erase vertex in min bucket
        int count = cpu_instance.bucket_set_to_array(min, cpu_instance.vertex_buf_ptr);
        //printf("min: %d  count: %d\n", min,count);

	    //set v set to zero, clear result buffer
	    //deploy vertex set to GPU
	    CUDA_SAFE_CALL(hipMemcpyAsync(cpu_instance.gpu_vertex_buf,cpu_instance.vertex_buf_ptr,
				vertex_buf_size,hipMemcpyHostToDevice));




        //call cuda function
        relax_all<<<num_block,num_threads>>>(cpu_instance.gpu_vertex_buf,cpu_instance.gpu_used_result_buf,
               cpu_instance.gpu_vertex,cpu_instance.gpu_edge);

        //CUT_CHECK_ERROR("Kernel execution failed\n");
        //get the result back
        //CUDA_SAFE_CALL(hipDeviceSynchronize());


        verify_result<<<1,NUM_BLOCK>>>(cpu_instance.gpu_vertex,cpu_instance.gpu_used_result_buf);
        //CUDA_SAFE_CALL(hipDeviceSynchronize());

        gettimeofday(&cpu_instance.start,NULL);
        CUDA_SAFE_CALL(hipMemcpyAsync(cpu_instance.gpu_result_buf,cpu_instance.gpu_used_result_buf,
				result_size,hipMemcpyDeviceToHost));


        gettimeofday(&cpu_instance.end,NULL);
        relax_time+=(cpu_instance.end.tv_sec*1000000 + cpu_instance.end.tv_usec)-(cpu_instance.start.tv_sec*1000000+cpu_instance.start.tv_usec);

        //get the result from gpu
        parse_result(cpu_instance);
       
    }
    get_result<<<1,1>>>(cpu_instance.gpu_vertex,cpu_instance.dest,cpu_instance.src);
    printf("over\n");
}

cpu::cpu(char* filepath, int src_p, int dest_p){
    init_memory(filepath);
    delta = 0x1fff;
    src = src_p;
    global_vertex[src].dist = 0;
    dest = dest_p;
    init_all_bucket();
}

cpu::~cpu(){

    free(global_vertex);
    free(global_edge);
    free(gpu_result_buf);
    free(vertex_buf_ptr);
    CUDA_SAFE_CALL(hipFree(gpu_vertex));
    CUDA_SAFE_CALL(hipFree(gpu_edge));
    CUDA_SAFE_CALL(hipFree(gpu_vertex_buf));
    CUDA_SAFE_CALL(hipFree(gpu_used_result_buf));
/*
    //mapped memory
    CUDA_SAFE_CALL(hipHostFree(vertex_buf_ptr));
    for(int i=0;i<NUM_BLOCK;i++)
        CUDA_SAFE_CALL(hipHostFree(gpu_result_buf[i]));
*/
}

int cpu::init_graph(){
  int i;
 //vertex size and edge size
  global_vertex =(struct vertex*) malloc((vertex_size+2)*sizeof(struct vertex));
  global_edge = (struct edge*)malloc(edges_size*sizeof(struct edge));

  //init vertex
  for(i=0;i<vertex_size+2;i++){
    global_vertex[i].edge_index =0;
    global_vertex[i].dist = MAX_DISTANCE;
    global_vertex[i].pre_vertex = -1;
  }

  graph_init=1;
  return 0;
}

int cpu::init_memory(char* filepath){
  char string[256];

  FILE* fp = fopen(filepath,"r");
  if(fp==NULL)
    return -1;

  //init edge, file staff
  while(fgets(string,256,fp)!=NULL){
    static char sign;
    //assumption1: the node start at index 1
    static int src,dest,dist,cur_v=0,cur_edge=0;
    //get the sign of the file line
    sscanf(string,"%c",&sign);

    //the line describe the edge
    if(sign=='a'){
      if(!graph_init)
	return -2;
      if(cur_edge>edges_size)
	return -4;
      sscanf(string,"%c\t%d\t%d\t%d",&sign,&src,&dest,&dist);
      dprintf("edge from:%d to:%d dist:%d\n",src,dest,dist);
      //add the edge to the edge list
      global_edge[cur_edge].des_v=dest;
      global_edge[cur_edge].distance=dist;
      cur_edge++;


      if(cur_v!=src){
	//assumption2: sorted vertex and there is no isolated vertex
	if(cur_v==src-1){
	  //global_vertex[src].edges=&global_edge[cur_edge-1];
	  global_vertex[src].edge_index=cur_edge-1;	
	  cur_v=src;
	}
	else
	  return -3;
      }
    }
    //the line describe the size of graph
    else if(sign=='p'){ 
      sscanf(string,"%c\tsp\t%d\t%d",&sign,&src,&dest);
      vertex_size = src;
      edges_size = dest;
      #ifdef DEBUG
      dprintf("GOT the size of graph, vertex:%d edge:%d\n",vertex_size,edges_size);
      #endif
      init_graph();
    }
  }

  fclose(fp);
  dprintf("end of init\n");
  //copy to GPU
 /* CUDA_SAFE_CALL(hipMalloc((void **)&gpu_vertex,(vertex_size+2)*sizeof(struct vertex)));
  CUDA_SAFE_CALL(hipMalloc((void**)&gpu_edge,edges_size*sizeof(struct edge)));
  CUDA_SAFE_CALL(hipMemcpy(gpu_vertex,global_vertex,(vertex_size+2)*sizeof(struct vertex),hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(gpu_edge,global_edge,edges_size*sizeof(struct edge)));
  */
  return 0;
}

int cpu::init_all_bucket(){
    dprintf("insert src : %d\n", src);
    bucket_array[0].insert(src);
}

int cpu::is_all_bucket_empty(){
    return min_no_empty_bucket()==-1;
}

int cpu::min_no_empty_bucket(){
    for(int i=0;i<MAX_BUKET_NUM;i++){
        if(!bucket_array[i].empty()){
              return i;
        }
    }
    return -1;
}

//copy to buffer and erase
int cpu::bucket_set_to_array(int index, int* array){
    int count = 0;
    std::set<int>::iterator it = bucket_array[index].begin();
    for(;it!=bucket_array[index].end();it++){
            array[count]=*it;
	    bucket_array[index].erase(it);
            count++;
	    if(count>V_BUF_SIZE){
//	    if(count>NUM_BLOCK){	
//		printf("oops!\n");
		return count;
	    }
    }
    return count;
}


int main(int argc, char **argv){
    int src_p = 1;
    int dest_p = 6;
    if(argc != 3){
        printf("Need two arguments for source and destination!\nDefault Source Point:%d\nDefault Destination Point:%d\n", src_p, dest_p);
    }
    else{
        src_p = atoi(argv[1]);
        dest_p = atoi(argv[2]);
        printf("Source Point:%d\nDestination Point:%d\n", src_p, dest_p);
    }
    
    struct timeval start,end;


    cpu cpu_instance("USA.gr", src_p, dest_p);

    hipSetDevice(cutGetMaxGflopsDeviceId());

    gpu_memory_prep(cpu_instance);

    gettimeofday(&start,NULL);
    cal_shortest_path(cpu_instance);
    gettimeofday(&end,NULL);
    printf("time cost: %d ms\n",((end.tv_sec*1000000+end.tv_usec)-(start.tv_sec*1000000+start.tv_usec))/1000);
    printf("relax time cost: %d ms\n",relax_time/1000);
    //CUT_EXIT();
}
